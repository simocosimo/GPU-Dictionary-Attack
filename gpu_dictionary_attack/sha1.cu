/*************************** HEADER FILES ***************************/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <memory.h>
#include <iostream>
#include <string>
#include <fstream>

/****************************** MACROS ******************************/
#define SHA1_BLOCK_SIZE 20              // SHA1 outputs a 20 byte digest
#define THREADS 256
#define PASSWORD_MAX_LENGTH	50
#define ALLOCATION_MAX_SIZE 75000000

/**************************** DATA TYPES ****************************/
typedef struct {
	unsigned char data[64];
	unsigned int datalen;
	unsigned long long bitlen;
	unsigned int state[5];
	unsigned int k[4];
} CUDA_SHA1_CTX;

// Struct needed to pass information to device
typedef struct {
	unsigned char word[PASSWORD_MAX_LENGTH];
	unsigned int len;
} PWD_INFO;

/****************************** MACROS ******************************/
#ifndef ROTLEFT
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#endif

// Macro to detect CUDA errors when calling cuda api functions
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*********************** FUNCTION DEFINITIONS ***********************/
__device__  __forceinline__ void cuda_sha1_transform(CUDA_SHA1_CTX *ctx, const unsigned char data[])
{
	unsigned int a, b, c, d, e, i, j, t, m[80];

	// create a 32-bit chunk
	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) + (data[j + 1] << 16) + (data[j + 2] << 8) + (data[j + 3]);

	// Extend the chunck up to an 80 bit one
	// From bit 16 to 79, execute this specific modification
	for ( ; i < 80; ++i) {
		m[i] = (m[i - 3] ^ m[i - 8] ^ m[i - 14] ^ m[i - 16]);
		m[i] = (m[i] << 1) | (m[i] >> 31);
	}

	// Take the state (from init or previous chunk calculation)
	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];

	for (i = 0; i < 20; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (~b & d)) + e + ctx->k[0] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for ( ; i < 40; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + ctx->k[1] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for ( ; i < 60; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (b & d) ^ (c & d))  + e + ctx->k[2] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for ( ; i < 80; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + ctx->k[3] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}

	// Update the state with current chunk calculation
	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
}

__device__ void cuda_sha1_init(CUDA_SHA1_CTX *ctx) {
	// Setting up the initial word and the k variable
	// These constants will be useful for future computation
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x67452301;
	ctx->state[1] = 0xEFCDAB89;
	ctx->state[2] = 0x98BADCFE;
	ctx->state[3] = 0x10325476;
	ctx->state[4] = 0xc3d2e1f0;
	ctx->k[0] = 0x5a827999;
	ctx->k[1] = 0x6ed9eba1;
	ctx->k[2] = 0x8f1bbcdc;
	ctx->k[3] = 0xca62c1d6;
}

__device__ void cuda_sha1_update(CUDA_SHA1_CTX *ctx, const unsigned char data[], size_t len) {
	size_t i;

	for (i = 0; i < len; ++i) {
		// copy the password in the data field
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;

		// If 64 bytes are reached, we have reached a block (512 bit)
		if (ctx->datalen == 64) {
			// Transform the copied content
			cuda_sha1_transform(ctx, ctx->data);

			// Update the indices to get ready for next block
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void cuda_sha1_final(CUDA_SHA1_CTX *ctx, unsigned char hash[]) {
	
	unsigned int i;
	i = ctx->datalen;

	// Pad whatever data is left in the buffer since everything
	// needs to be a 512 bit multiple
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	} else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		cuda_sha1_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	cuda_sha1_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and MD uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
	}
}

__device__ __host__ void makedigits(unsigned char x, unsigned char (&digits)[2]) {
    unsigned char d0 = x / 16;
    digits[1] = x - d0 * 16;
    unsigned char d1 = d0 / 16;
    digits[0] = d0 - d1 * 16;
}

__device__ __host__ void makehex(unsigned char (&digits)[2], char (&hex)[2]) {
    for(int i=0; i<2; ++i) {
        if (digits[i] < 10) {
            hex[i] = '0' + digits[i];
    } else {
            hex[i] = 'a' + (digits[i] - 10);
        }
    }
}

// Main kernel
// The idea is the following: 256 threads per block are spawned. Each thread will compute the hash
// of a single password. It then checks if the resulting hash is equal to the target one.
// If it is, it chagnes the match variable to the index of the found password, otherwise it will
// just end computing and stops.
__global__ void kernel_sha1_hash_BUFFER(PWD_INFO *indata, 
			unsigned int total_threads, 
			unsigned int maxBlock, 
			unsigned char *target, 
			int *match)
{
	// Info about thread/block
	unsigned int bId = blockIdx.x;
	unsigned int tId = threadIdx.x;
	unsigned int thread = bId * blockDim.x + tId;

	// Using register to store the result of the hash
	unsigned char cache[SHA1_BLOCK_SIZE];

	// Stopping condition: # of passwords could not be an exact multiple of 256
	// So the last block needs to stop all threads that which id is higher than the
	// modulo of total_threads by 256 (ex. if last block needs to calculate 64 passwords
	// only the first 64 threads will work, other will exit).
	// With this, I also try to stop other threads to start if a result has already 
	// been found (match is no more == -1).
	if ((bId == maxBlock && tId >= total_threads % THREADS) || *match != -1)
	{
		return;
	}

	// Each thread gets a pwd and calculate the hash on it
	// save the result in the cache variable
	unsigned char* in = indata[thread].word;
	unsigned int len = indata[thread].len;
	CUDA_SHA1_CTX ctx;
	cuda_sha1_init(&ctx);
	cuda_sha1_update(&ctx, in, len);
	cuda_sha1_final(&ctx, cache);

	// Here there is the conversion from an array of unsigned char
	// to a hex string representation. When a single byte is 
	// converted, we check with the target string. If the conversion
	// is different we immediatly stop computing since this is not
	// the hash we're looking for
	for(int i = 0; i < SHA1_BLOCK_SIZE; i++) {
		unsigned char val = cache[i];
		unsigned char d[2];
		char h[2];
		makedigits(val, d);
		makehex(d, h);
		if(target[2*i] != h[0] || target[2*i+1] != h[1]) {
			return;
		}
	}

	// The thread that will "survive" the cycle before will
	// set the match variable with its thread information.
	// Host can now retrieve the clear text password
	*match = thread;
}

void mcm_cuda_sha1_hash_batch_BUFFER(PWD_INFO *cuda_indata, 
			unsigned int buffer_len,
			unsigned char *target,
			int *match
			)
{
	// Threads will be 256 per block (experiments show that it's the better option)
	// The calculation of the blocks will be done based on the amount of passwords to
	// be checked against the target
	unsigned int thread = THREADS;
	unsigned int block = (buffer_len + thread - 1) / thread;
	
	// Kernel call
	kernel_sha1_hash_BUFFER<<< block, thread >>>(cuda_indata, buffer_len, block - 1, target, match);

	// Synchronization
	gpuErrchk( hipDeviceSynchronize());

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error cuda sha1 hash: %s \n", hipGetErrorString(error));
	}

}

// 1 - Dictionary filename
// 2 - Dictionary dimension
// 3 - Target
int main(int argc, char **argv) {
	char *filename = argv[1];
    std::ifstream passwords(filename);
	std::string line;

	PWD_INFO *cuda_indata;
	PWD_INFO *local_indata;

	int db_size = std::stoi(argv[2]);
	char *target = argv[3];
	unsigned char *d_target;
	
	int h_match = -1;
	int *d_match;

	unsigned int alloc_size = (db_size > ALLOCATION_MAX_SIZE ? ALLOCATION_MAX_SIZE : db_size);

	gpuErrchk( hipMalloc(&cuda_indata, alloc_size * sizeof(PWD_INFO)));
	local_indata = (PWD_INFO *)malloc(alloc_size * sizeof(PWD_INFO));
	gpuErrchk( hipMalloc(&d_target, SHA1_BLOCK_SIZE * 2 * sizeof(unsigned char)));
	gpuErrchk( hipMalloc(&d_match, sizeof(int)));

	// Reading the file and saving the clear text password and its lenght in a
	// PWD_INFO struct. This will then be copied to the device.
	int size = 0;
	while(std::getline(passwords, line)) {
		local_indata[size].len = strlen(line.c_str());
		strcpy((char *)local_indata[size++].word, line.c_str());
	}

	hipEvent_t start, stop, startKern, stopKern;
	float elapsedTimeKern, elapsedTime;

	// Start measuring time for memory + kernel
	gpuErrchk( hipEventCreate( &start ) );
	gpuErrchk( hipEventCreate( &stop ) );
	gpuErrchk( hipEventRecord( start, 0 ) );

	// Copying data from host to memory: array of PWD_INFO, target hash 
	// and the variable for the index of the (possibly) found password
	gpuErrchk( hipMemcpy(cuda_indata, local_indata, alloc_size * sizeof(PWD_INFO), hipMemcpyHostToDevice));
	gpuErrchk( hipMemcpy(d_target, target, SHA1_BLOCK_SIZE * 2 * sizeof(unsigned char), hipMemcpyHostToDevice));
	gpuErrchk( hipMemcpy(d_match, &h_match, sizeof(int), hipMemcpyHostToDevice));

	// Start measuring time for just kernel
	gpuErrchk( hipEventCreate( &startKern ) );
	gpuErrchk( hipEventCreate( &stopKern ) );
	gpuErrchk( hipEventRecord( startKern, 0 ) );

	// Wrapper that internally calls the kernel
	mcm_cuda_sha1_hash_batch_BUFFER(cuda_indata, size, d_target, d_match);

	// Stopping measuring time for just kernel and printing results
	gpuErrchk( hipEventRecord( stopKern, 0 ) );
	gpuErrchk( hipEventSynchronize( stopKern ) );
	gpuErrchk( hipEventElapsedTime( &elapsedTimeKern, startKern, stopKern ) );
	printf( "Kernel timing: %3.1f ms\n", elapsedTimeKern );

	// Copying index result from device to host. No need to copy back the whole 
	// PWD_INFO array since it is no more useful to the host (or better, it is unchanged)
	gpuErrchk( hipMemcpy(&h_match, d_match, sizeof(int), hipMemcpyDeviceToHost));

	// Stopping measuring time for just kernel and printing results
	gpuErrchk( hipEventRecord( stop, 0 ) );
	gpuErrchk( hipEventSynchronize( stop ) );
	gpuErrchk( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf( "Kernel + memory timing: %3.1f ms\n", elapsedTime );

	// Destroying event handlers
	gpuErrchk( hipEventDestroy( startKern ) );
	gpuErrchk( hipEventDestroy( stopKern ) );
	gpuErrchk( hipEventDestroy( start ) );
	gpuErrchk( hipEventDestroy( stop ) );

	// Displaying the result
    if(h_match == -1) {
        std::cout << "Password not in database" << std::endl;
    } else {
	    std::cout << "Password is " << local_indata[h_match].word << std::endl;
    }

	// Freeing all the pointers (device and host) dynamically allocated
	gpuErrchk( hipFree(cuda_indata));
	gpuErrchk( hipFree(d_match));
	gpuErrchk( hipFree(d_target));
	free(local_indata);

    return 0;
}